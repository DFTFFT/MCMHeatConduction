
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <sys/time.h>

#define NX 19				//the number of internal nodes in the x direction
#define NY 19				//the number of internal nodes in the y direction
#define SIZE (NX+2)*(NY+2)		//total number of nodes (including boundary nodes)

#define MIN_ITER 10000

__device__ double Tpre[SIZE];		//temperature of the previous timestep

__global__ void setup_kernel(hiprandState *state, double *dev_T)
{
	int i, j;
	int NThd_x;
	int idx;

	i = threadIdx.x + blockIdx.x*blockDim.x;
	j = threadIdx.y + blockIdx.y*blockDim.y;
	NThd_x = blockDim.x*gridDim.x;
	idx = i + j*NThd_x;

	hiprand_init(1234, idx, 0, &state[idx]);

	for(i=0; i<SIZE; i++)
		Tpre[i] = dev_T[i];
}



__global__ void kernel(double *T, double *para, hiprandState *state)
{
	double LX = para[0];		//length in x direction, m
	double LY = para[1];		//length in y direction, m
	double k = para[2];		//conductivity, W(m-K)
	double Q = para[3];		//internal heat source, W/m2
	double rho = para[4];		//density of the material, kg/m3
	double cp = para[5];		//specific heat capacity of the material, J/(kg-K)
	double dt = para[6];		//time step, sec

	int i;				//index in the x direction
	int j;				//index in the y direction
	int NThd_x;			//number of threads in the x direction
	int idx;			//index of the temperature nodes in 1D array	

	double r;			//random number
	double Tsum = 0.0;		//accumulated temperature
	double Told = 0.0;		//old value of the node's temperature
	double TH[] = {0.0, 0.0, 0.0,
		0.0};			//threshold value for random walking
	double q = 0.0;			//source term
	double dx = LX/(NX+1);		//spacial interval length in x direction, m
	double dy = LY/(NY+1);		//spacial interval length in y direction, m
	int pos_x = -1;			//current position of the node in the x direction
	int pos_y = -1;			//current position of the node in the y direction
	int flag = 1;			//flag for the outmost iteration
	int iflag = 0;			//flag for internal iteration
	int sflag = 0;			//flag for the stationary state
	int count = 0;			//number of iterations
	double err = 0.0;		//absolute error
	double const EPS = 1.0E-3;	//error tolerance

	double a[4];			//temporary storage the coefficients
	//
	NThd_x = blockDim.x*gridDim.x;
	j = threadIdx.x + blockIdx.x*blockDim.x;
	i = threadIdx.y + blockIdx.y*blockDim.y;

	idx = j + NThd_x*i;
	//save the result of the previous timestep
	Tpre[idx] = T[idx];
	//
	hiprandState localState = state[idx];
	//
	a[0] = rho*cp/dt;
	a[1] = k/(dx*dx);
	a[2] = k/(dy*dy);
	a[3] = a[0]+2.0*a[1]+2.0*a[2];

	TH[0] = a[1]/a[3];		//threshold for TW0
	TH[1] = TH[0]+a[1]/a[3];	//threshold for TE0
	TH[2] = TH[1]+a[2]/a[3];	//threshold for TS0
	TH[3] = TH[2]+a[2]/a[3];	//threshold for TN0
	q = Q/a[3];			//normalized source term 
	//
	if(i>0 && i<(NY+1) && j>0 && j<(NX+1))
	{
		while(flag)
		{	
			count++;
			Told = T[idx];
			Tsum = T[idx]*(count-1);
			pos_x = j;
			pos_y = i;
			iflag = 0;
			sflag = 0;
			//
			while(!iflag)
			{
				r = hiprand_uniform(&localState);
				//
				if(r<TH[0])
					//move to west
					pos_x--;
				else if(r<TH[1])
					//move to east
					pos_x++;
				else if(r<TH[2])
					//move to south
					pos_y--;
				else if(r<TH[3])
					//move to north
					pos_y++;
				else
				{
					pos_x += 0;
					pos_y += 0;
					sflag = 1;
				}
				//
				Tsum += q;
				if(sflag)
				{
					iflag = 1;
					Tsum += Tpre[pos_x+NThd_x*pos_y];
				}
				else if(pos_x == 0 || pos_x == NX+1 || pos_y == 0 || pos_y == NY+1)
				{
					iflag = 1;
					Tsum += T[pos_x+NThd_x*pos_y];
				}
			}
			//
			T[idx] = Tsum/count;
			err = fabs(T[idx]-Told);
			if(err<EPS && count>MIN_ITER)
			{
				flag = 0;
			}
			//state[idx] = localState;
		}
		//
		state[idx] = localState;
	}
}

int main()
{
	double const TW = 200.0;		//west boundary temperature, C
	double const TE = 150.0;		//east boundary temperature, C
	double const TS = 100.0;		//south boundary temperature, C
	double const TN = 50.0;			//north boundary temperature, C

	double const LX = 0.20;			//length in the x direction, m
	double const LY = 0.15;			//length in the y direction, m
	double const k = 385.0;			//conductivity, W(m-K)
	double const Q = 0.0;			//internal heat source, W/m2
	double const rho = 8.96E3;		//density, kg/m3
	double const cp = 3.85E2;		//specific capacity, J/(kg-K)

	int const NStep = 100;			//number of timestep
	double cal_time = 0.0;			//current time, sec
	double end_time = 10.0;			//end time
	double dt = end_time/NStep;		//time step, sec
	
	int const blocksize = 1;		//number of threads in each block
	dim3 dimBlock(blocksize, blocksize);
	dim3 dimGrid((NY+2+blocksize-1)/blocksize, (NX+2+blocksize-1)/blocksize);

	int i, j, t;
	double **Tfield;			//The field of temperature (2D)
	double *T;				//linearized temperature stored in 1D array
	double *dev_T;
	double Tinit = 0.25*(TW+TE+TS+TN);	//initial temperture, C
	double x;				//x-coordinate of node, m
	double y;				//y-coordinate of node, m
	double dx = LX/(NX+1);
	double dy = LY/(NY+1);
	
	double para[] 
		= {LX, LY, k, Q, 
		rho, cp, dt};			//parameters for solving
	double *dev_para;
	int num_para = sizeof(para)/sizeof(double);

	hiprandState *devStates;
	
	time_t startTime, endTime;
	struct timeval start, end;
	
	FILE *fp;				//file pointer
	//
	fp = fopen("Result.txt", "w");
	if(fp == NULL)
	{
		printf("Fail to open the result.txt file!");
	}
	//
	Tfield = (double**)malloc((NY+2)*sizeof(double*));
	for(i=0; i<NY+2; i++)
		Tfield[i] = (double*)malloc((NX+2)*sizeof(double));

	T = (double*)malloc(SIZE*sizeof(double));

	hipMalloc((void**)&dev_T, SIZE*sizeof(double));
	hipMalloc((void**)&devStates, SIZE*sizeof(hiprandState));
	hipMalloc((void**)&dev_para, num_para*sizeof(double));
	//
	for(i=0; i<NY+2; i++)
	{
		for(j=0; j<NX+2; j++)
		{
			if(i == 0)
				Tfield[i][j] = TS;
			else if(i == NY+1)
				Tfield[i][j] = TN;
			else if(j == 0)
				Tfield[i][j] = TW;
			else if(j == NX+1)
				Tfield[i][j] = TE;
			else
				Tfield[i][j] = Tinit;
			//
			T[j+(NX+2)*i] = Tfield[i][j];
		}
	}
	//
	hipMemcpy(dev_T, T, SIZE*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_para, para, num_para*sizeof(double), hipMemcpyHostToDevice);
	//
	setup_kernel<<<dimGrid,dimBlock>>>(devStates, dev_T);
	//
	startTime = time(NULL);
	gettimeofday(&start, NULL);
	//
	for(t=0; t<NStep; t++)
	{
		cal_time += dt;
		//
		kernel<<<dimGrid, dimBlock>>>(dev_T, dev_para, devStates);
		//
		//cudaThreadSynchronize();
		hipDeviceSynchronize();
		//	
		hipMemcpy(T, dev_T, SIZE*sizeof(double), hipMemcpyDeviceToHost);
		//
		fprintf(fp, "Results of time = %f:\n", cal_time);
		for(i=0; i<NY+2; i++)
		{
			for(j=0; j<NX+2; j++)
			{
				x = j*dx;
				y = i*dy;
				Tfield[i][j] = T[j+(NX+2)*i];
				fprintf(fp, "%f\t%f\t%f\n", x, y, Tfield[i][j]);
			}
		}
		fprintf(fp, "\n\n");
	}
	//
	endTime = time(NULL);
	gettimeofday(&end, NULL);
	printf("The calculation time is: %f seconds\n", difftime(endTime, startTime));
	double timelapse = (end.tv_sec-start.tv_sec) + (end.tv_usec-start.tv_usec)/1.0E6;
	printf("The time used for calculation is %f\n", timelapse);
	//
	hipFree(dev_T);
	hipFree(dev_para);
	fclose(fp);
	//
	return 0;
}

